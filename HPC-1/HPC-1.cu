
#include <hip/hip_runtime.h>
#include<iostream>
#include<stdio.h>
#include<math.h>
#include<stdlib.h>
#include<time.h>
#define N 2048
using namespace std;

__global__ void Max(int *a)
{
    int tid = threadIdx.x;
    int step_size = 1;
    int num_of_threads = blockDim.x;
    
    while(num_of_threads>0){
        if(tid<num_of_threads){
            int first = tid*step_size*2;
            int second = first+step_size;
            if(a[second]>a[first])
                a[first]=a[second];                  
        }
        step_size<<=1;
        num_of_threads>>=1;
    }
    
}
__global__ void Min(int *a)
{
    int tid = threadIdx.x;
    int step_size = 1;
    int num_of_threads = blockDim.x;
    
    while(num_of_threads>0){
        if(tid<num_of_threads){
            int first = tid*step_size*2;
            int second = first+step_size;
            if(a[second]<a[first])
                a[first]=a[second];                  
        }
        step_size<<=1;
        num_of_threads>>=1;
    }
}
__global__ void sum(int *a)
{
    int tid = threadIdx.x;
    int step_size = 1;
    int num_of_threads = blockDim.x;
    
    while(num_of_threads>0){
        if(tid<num_of_threads){
            int first = tid*step_size*2;
            int second = first+step_size;
            a[first]+=a[second];                  
        }
        step_size<<=1;
        num_of_threads>>=1;
    }
}
__global__ void mean_diff(float *a,float mean)
{
    a[threadIdx.x]-=mean;
    a[threadIdx.x]*=a[threadIdx.x];
    
}
__global__ void _std(float *a,int n)
{
    int tid = threadIdx.x;
    int step_size = 1;
    int num_of_threads = blockDim.x;
    
    while(num_of_threads>0){
        if(tid<num_of_threads){
            int first = tid*step_size*2;
            int second = first+step_size;
            a[first]+=a[second];                  
        }
        step_size<<=1;
        num_of_threads>>=1;
    }
    a[0]/=a[0]/n;
}

int main()
{
    int *a = (int *)malloc(N*sizeof(int));
    float *f = (float *)malloc(N*sizeof(float));
    //srand(time(0));
    for(int i=0;i<N;i++){
        a[i]=rand()%10;
        f[i]=float(a[i]);
    } 
    int *a_cuda;
    float *f_cuda;          
    int answer;
    
    hipMalloc((void **)&a_cuda,N*sizeof(int));
   
    //MAX
    hipMemcpy(a_cuda,a,N*sizeof(int),hipMemcpyHostToDevice);
    Max<<<1,N/2>>>(a_cuda);
    hipMemcpy(&answer,a_cuda,sizeof(int),hipMemcpyDeviceToHost);
    cout<<"Max : "<<answer<<endl;
    
    //MIN
    hipMemcpy(a_cuda,a,N*sizeof(int),hipMemcpyHostToDevice);
    Min<<<1,N/2>>>(a_cuda);
    hipMemcpy(&answer,a_cuda,sizeof(int),hipMemcpyDeviceToHost);
    cout<<"Min : "<<answer<<endl;
    
    //SUM
    hipMemcpy(a_cuda,a,N*sizeof(int),hipMemcpyHostToDevice);
    sum<<<1,N/2>>>(a_cuda);
    hipMemcpy(&answer,a_cuda,sizeof(int),hipMemcpyDeviceToHost);
    cout<<"Sum : "<<answer<<endl;
    
    //MEAN
    float mean = float(answer)/N;
    cout<<"Mean : "<<mean<<endl;
              
    hipFree(a_cuda);
    hipMalloc((void **)&f_cuda,N*sizeof(float));
    float result;
    //STD
    hipMemcpy(f_cuda,f,N*sizeof(float),hipMemcpyHostToDevice);          
    mean_diff<<<1,N>>>(f_cuda,mean);        
    _std<<<1,N/2>>>(f_cuda,N);
    hipMemcpy(&result,f_cuda,sizeof(float),hipMemcpyDeviceToHost);
    cout<<"Variance : "<<result<<endl;
    cout<<"Std. Dev. : "<<sqrt(result)<<endl;
              
    hipFree(f_cuda);    
    cout<<endl;
    return 0;          
}