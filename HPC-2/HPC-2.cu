
#include <hip/hip_runtime.h>
#include<iostream>
#include<stdio.h>
#include<math.h>
#include<stdlib.h>
#include<time.h>
#define N 4
#define t_per_block 2
using namespace std;

void random_ints(int *vec, int size){
    for(int i=0; i<size; i++)
        vec[i] = i;
}

void random_ints_mat(int *mat, int size){
    int k=0;
    for(int i=0; i<size; i++){
        for(int j=0; j<size; j++){
            mat[i*size+j] = k++;
        }
    }
}

__global__ void addV(int *a,int *b,int *c)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    c[index] = a[index]+b[index];   
}
__global__ void MulMatVec(int *vec,int *mat,int *out_vec, int n)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int sum=0;
    for(int i=0;i<n;i++)
        sum+=vec[i]*mat[(i*n)+index];
    out_vec[index] = sum;
    
}
__global__ void MulMatMat(int *a, int *b, int *c, int n)
{
    int row_index = threadIdx.y + blockIdx.y * blockDim.y;
    int col_index = threadIdx.x + blockIdx.x * blockDim.x;
    int sum=0;
    for(int i=0;i<n;i++)
        sum+=a[(row_index*n)+i] * b[(i*n)+col_index];
    c[row_index*n + col_index]=sum;
}

int main()
{
    // ----------- Vector Addition ----------------
    int *a, *b, *c; //Vectors on host
    int *d_a, *d_b, *d_c; //Vectors on device
    int v_size = N*sizeof(int);
    
    a = (int *)malloc(v_size);
    b = (int *)malloc(v_size);
    c = (int *)malloc(v_size);
    
    hipMalloc((void **)&d_a, v_size);
    hipMalloc((void **)&d_b, v_size);
    hipMalloc((void **)&d_c, v_size);
    
    random_ints(a,N);
    random_ints(b,N);
    
    hipMemcpy(d_a, a, v_size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, v_size, hipMemcpyHostToDevice);
    
    addV<<<N/t_per_block, t_per_block>>>(d_a, d_b, d_c);
    
    hipMemcpy(c, d_c, v_size, hipMemcpyDeviceToHost);
   
    hipFree(d_a); hipFree(d_b); hipFree(d_c); 
    free(a); free(b); free(c); 
    
    
    // ----------- Vector Matrix Multiplication ----------- 
    int *vec, *mat, *out_vec;
    vec = (int *)malloc(N*sizeof(int));
    mat = (int *)malloc(N*N*sizeof(int));
    out_vec = (int *)malloc(N*sizeof(int));
    
    int *d_vec, *d_mat, *d_out_vec;
    hipMalloc((void **)&d_vec, N*sizeof(int));
    hipMalloc((void **)&d_mat, N*N*sizeof(int));
    hipMalloc((void **)&d_out_vec, N*sizeof(int));
    
    random_ints(vec, N);
    random_ints(mat, N*N);
    
    hipMemcpy(d_vec, vec, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_mat, mat, N*N*sizeof(int), hipMemcpyHostToDevice);
    
    MulMatVec<<<N/t_per_block, t_per_block>>>(d_vec, d_mat, d_out_vec, N);
    
    hipMemcpy(out_vec, d_out_vec, N*sizeof(int), hipMemcpyDeviceToHost);
  
    hipFree(d_vec); hipFree(d_mat); hipFree(d_out_vec); 
    free(vec); free(mat); free(out_vec); 

    // ----------- Matrix Multiplication ----------- 
    int *mat_a, *mat_b, *mat_c;
    int *d_mat_a, *d_mat_b, *d_mat_c;
    
    mat_a = (int *)malloc(N*N*sizeof(int));
    mat_b = (int *)malloc(N*N*sizeof(int));
    mat_c = (int *)malloc(N*N*sizeof(int));

    hipMalloc((void **)&d_mat_a, N*N*sizeof(int));
    hipMalloc((void **)&d_mat_b, N*N*sizeof(int));
    hipMalloc((void **)&d_mat_c, N*N*sizeof(int));
    
    random_ints_mat(mat_a, N);
    random_ints_mat(mat_b, N);
    
    hipMemcpy(d_mat_a, mat_a, N*N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_mat_b, mat_b, N*N*sizeof(int), hipMemcpyHostToDevice);
    
    dim3 threadsPerBlock(2,2);
    dim3 blocksPerGrid(N/threadsPerBlock.x,N/threadsPerBlock.y);
    
    MulMatMat<<<blocksPerGrid,threadsPerBlock>>>(d_mat_a, d_mat_b, d_mat_c, N);
    
    hipMemcpy(mat_c, d_mat_c, N*N*sizeof(int), hipMemcpyDeviceToHost);
    
    for(int i=0;i<N;i++){
        for(int j=0;j<N;j++){
            printf("%d ", mat_c[i*N + j]);
        }
        printf("\\n");
    }
    
    hipFree(d_mat_a); hipFree(d_mat_b); hipFree(d_mat_c); 
    free(mat_a); free(mat_b); free(mat_c);     
    
    return 0;       
}